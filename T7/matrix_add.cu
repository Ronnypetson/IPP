
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void add(int *a, int *b, int *c){
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    c[index] = a[index] + b[index];
}

int main()
{
    int *A, *B, *C;
    int *d_A, *d_B, *d_C;
    int i, j;

    //Input
    int linhas, colunas;

    scanf("%d", &linhas);
    scanf("%d", &colunas);

    //Definindo tamanho dos arrays que representarão as matrizes
    int N = linhas*colunas;
    int size = sizeof(int)*N;

    //Alocando memória na GPU
    hipMalloc((void **)&d_A,size);
    hipMalloc((void **)&d_B,size);
    hipMalloc((void **)&d_C,size);

    //Alocando memória na CPU
    A = (int *)malloc(size);
    B = (int *)malloc(size);
    C = (int *)malloc(size);

    //Inicializar
    for(i = 0; i < linhas; i++){
        for(j = 0; j < colunas; j++){
            A[i*colunas+j] =  B[i*colunas+j] = i+j;
            //printf("%d ",A[i*colunas+j]);
        }
        //printf("\n");
    }

    //Transferir para a memória da GPU
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);

    //Computacao que deverá ser movida para a GPU
    //Lembrar que é necessário usar mapeamento 2D (visto em aula) 
    //for(i=0; i < linhas; i++){
    //    for(j = 0; j < colunas; j++){
    //        C[i*colunas+j] = A[i*colunas+j] + B[i*colunas+j];
    //    }
    //}
    // Número de blocos = Número de linhas
    // threads por bloco = número de colunas
    add<<<linhas,colunas>>>(d_A,d_B,d_C);
    hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

    long long int somador=0;
    //Manter esta computação na CPU
    for(i = 0; i < linhas; i++){
        for(j = 0; j < colunas; j++){
            somador+=C[i*colunas+j];
            //printf("%d ",C[i*colunas+j]);
        }
        //printf("\n");
    }
    
    printf("%lli\n", somador);

    free(A); free(B); free(C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

