
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define THREADS_POR_BLOCO 1024
#define DIM_BLOCO 32
#define DIM_GRID 1024

__global__ void add(int *a, int *b, int *c, int N){
    //int i = threadIdx.y + blockIdx.y*blockDim.y;
    //int j = threadIdx.x + blockIdx.x*blockDim.x;
    int index = 
        DIM_BLOCO*DIM_BLOCO*(DIM_GRID*blockIdx.x+blockIdx.y)+blockDim.y*threadIdx.x+threadIdx.y;
    if(index < N)
        c[index] = a[index] + b[index];
}

int main()
{
    int *A, *B, *C;
    int *d_A, *d_B, *d_C;
    int i, j;

    //Input
    int linhas, colunas;

    scanf("%d", &linhas);
    scanf("%d", &colunas);

    //Definindo tamanho dos arrays que representarão as matrizes
    int N = linhas*colunas;
    int size = sizeof(int)*N;

    //Alocando memória na GPU
    hipMalloc((void **)&d_A,size);
    hipMalloc((void **)&d_B,size);
    hipMalloc((void **)&d_C,size);

    //Alocando memória na CPU
    A = (int *)malloc(size);
    B = (int *)malloc(size);
    C = (int *)malloc(size);

    //Inicializar
    for(i = 0; i < linhas; i++){
        for(j = 0; j < colunas; j++){
            A[i*colunas+j] =  B[i*colunas+j] = i+j;
            //printf("%d ",A[i*colunas+j]);
        }
        //printf("\n");
    }

    //Transferir para a memória da GPU
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);

    //Computacao que deverá ser movida para a GPU
    // Número de blocos = Número de linhas
    // threads por bloco = número de colunas
    dim3 dimGrid(DIM_GRID,DIM_GRID);
    dim3 dimBlock(DIM_BLOCO,DIM_BLOCO);
    add<<<dimGrid,dimBlock>>>(d_A,d_B,d_C,N);
    //add<<<(N+THREADS_POR_BLOCO-1)/THREADS_POR_BLOCO,THREADS_POR_BLOCO>>>(d_A,d_B,d_C,N);
    hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

    long long int somador=0;
    //Manter esta computação na CPU
    for(i = 0; i < linhas; i++){
        for(j = 0; j < colunas; j++){
            somador+=C[i*colunas+j];
            //printf("%d ",C[i*colunas+j]);
        }
        //printf("\n");
    }
    
    printf("%lli\n", somador);

    free(A); free(B); free(C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
}
