
#include <hip/hip_runtime.h>
#include <cstdio>
#define N 10

using namespace std;

__global__ void add(int *a, int *b, int *c){
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_ints(int* a){
    int i;
    for(i = 0; i < N; i++)
        a[i] = i*i;
}

int main(void){
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N*sizeof(int);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    a = (int *)malloc(size); random_ints(a);
    b = (int *)malloc(size); random_ints(b);
    c = (int *)malloc(size);

    //for(int i = 0; i < N; i++)
    //    printf("%d %d\n",a[i],b[i]);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<N,1>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++)
        printf("%d + %d = %d\n",a[i],b[i],c[i]);

    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}
