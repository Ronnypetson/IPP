
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#define COMMENT "Histogram_GPU"
#define RGB_COMPONENT_COLOR 255
#define DIM_BLOCO 32
#define DIM_GRID 1960 // 1960*1960*1024

typedef struct {
	unsigned char red, green, blue;
} PPMPixel;

typedef struct {
	int x, y;
	PPMPixel *data;
} PPMImage;

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}

static PPMImage *readPPM(const char *filename) {
	char buff[16];
	PPMImage *img;
	FILE *fp;
	int c, rgb_comp_color;
	fp = fopen(filename, "rb");
	if (!fp) {
		fprintf(stderr, "Unable to open file '%s'\n", filename);
		exit(1);
	}

	if (!fgets(buff, sizeof(buff), fp)) {
		perror(filename);
		exit(1);
	}

	if (buff[0] != 'P' || buff[1] != '6') {
		fprintf(stderr, "Invalid image format (must be 'P6')\n");
		exit(1);
	}

	img = (PPMImage *) malloc(sizeof(PPMImage));
	if (!img) {
		fprintf(stderr, "Unable to allocate memory\n");
		exit(1);
	}

	c = getc(fp);
	while (c == '#') {
		while (getc(fp) != '\n');
		c = getc(fp);
	}

	ungetc(c, fp);
	if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
		fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
		exit(1);
	}

	if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
		fprintf(stderr, "Invalid rgb component (error loading '%s')\n",
				filename);
		exit(1);
	}

	if (rgb_comp_color != RGB_COMPONENT_COLOR) {
		fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
		exit(1);
	}

	while (fgetc(fp) != '\n');
	img->data = (PPMPixel*) malloc(img->x * img->y * sizeof(PPMPixel));

	if (!img) {
		fprintf(stderr, "Unable to allocate memory\n");
		exit(1);
	}

	if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
		fprintf(stderr, "Error loading image '%s'\n", filename);
		exit(1);
	}

	fclose(fp);
	return img;
}

__global__ void count_hist(PPMPixel *data, float *h, unsigned int n_){
	// n_ é ó total de pixels
	// Uma thread para cada combinação (pixel,rgb), onde pixel = 0, 1, ..., n_-1 e rgb = 0, 1, ..., 63
	// Calcular índice da thread e os respectivos índices x, i, j, k, l
	// DIM_BLOCO = blockDim.x = blockDim.y
	// DIM_GRID = gridDim.x = gridDim.y
	unsigned int index = DIM_BLOCO*DIM_BLOCO*(DIM_GRID*blockIdx.x+blockIdx.y)+blockDim.y*threadIdx.x+threadIdx.y;
	unsigned int x = index/n_;
	unsigned int i = index%n_;
	unsigned int j = x/16;
	unsigned int k = (x-16*j)/4;
	unsigned int l = (x-16*j-4*k);
	if (index < 64*n_
		 && data[i].red == j
		 && data[i].green == k
		 && data[i].blue == l) {
		atomicAdd(&h[x],1.0); // o histograma é normalizado depois para evitar erro de precisão
	}
}

void Histogram(PPMImage *image, float *h) {
	int i, j, k, l, x, count;
	int rows, cols;
	unsigned int n = image->y * image->x;
	double t_start, t_end, t_cbuffer, t_offload_enviar, t_kernel, t_offload_receber;
	cols = image->x;
	rows = image->y;
	for (i = 0; i < n; i++) {
		image->data[i].red = floor((image->data[i].red * 4) / 256);
		image->data[i].blue = floor((image->data[i].blue * 4) / 256);
		image->data[i].green = floor((image->data[i].green * 4) / 256);
	}
	// Parte movida para a GPU
	unsigned int size = 3*sizeof(unsigned char)*n;
	PPMPixel *d_data;
	float *d_h;

	t_start = rtclock();
	hipMalloc((void **)&d_data,size);
	hipMalloc((void **)&d_h,64*sizeof(float));
	t_end = rtclock();
	t_cbuffer = t_end-t_start;
	
	t_start = rtclock();
	hipMemcpy(d_data,image->data,size,hipMemcpyHostToDevice);
	hipMemcpy(d_h,h,64*sizeof(float),hipMemcpyHostToDevice);
	t_end = rtclock();
	t_offload_enviar = t_end-t_start;
	
	dim3 dimGrid(DIM_GRID,DIM_GRID);
	dim3 dimBlock(DIM_BLOCO,DIM_BLOCO);

	t_start = rtclock();
	count_hist<<<dimGrid,dimBlock>>>(d_data,d_h,n);
	hipDeviceSynchronize();
	t_end = rtclock();
	t_kernel = t_end-t_start;

	t_start = rtclock();
	hipMemcpy(h,d_h,64*sizeof(float),hipMemcpyDeviceToHost);
	t_end = rtclock();
	t_offload_receber = t_end-t_start;

	hipFree(d_data); hipFree(d_h);
	for(i = 0; i < 64; i++)
		h[i] /= n;
	double t_total = t_cbuffer+t_offload_enviar+t_kernel+t_offload_receber;
	printf("%lf\t%lf\t%lf\t%lf\t%lf\n",t_cbuffer,t_offload_enviar,t_kernel,t_offload_receber,t_total);
}

int main(int argc, char *argv[]) {
	if( argc != 2 ) {
		printf("Too many or no one arguments supplied.\n");
	}

	double t_start, t_end;
	int i;
	char *filename = argv[1]; //Recebendo o arquivo!;
	
	//scanf("%s", filename);
	PPMImage *image = readPPM(filename);

	float *h = (float*)malloc(sizeof(float) * 64);

	//Inicializar h
	for(i=0; i < 64; i++) h[i] = 0.0;

	t_start = rtclock();
	Histogram(image, h);
	t_end = rtclock();

	for (i = 0; i < 64; i++){
		printf("%0.3f ", h[i]);
	}
	fprintf(stdout, "\n%0.6lfs\n", t_end - t_start);  
	free(h);
}

