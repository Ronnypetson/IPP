#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#define COMMENT "Histogram_GPU"
#define RGB_COMPONENT_COLOR 255
#define DIM_BLOCO 32
#define DIM_GRID 1960 // 1960*1960*1024

typedef struct {
	unsigned char red, green, blue;
} PPMPixel;

typedef struct {
	int x, y;
	PPMPixel *data;
} PPMImage;

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}

static PPMImage *readPPM(const char *filename) {
	char buff[16];
	PPMImage *img;
	FILE *fp;
	int c, rgb_comp_color;
	fp = fopen(filename, "rb");
	if (!fp) {
		fprintf(stderr, "Unable to open file '%s'\n", filename);
		exit(1);
	}

	if (!fgets(buff, sizeof(buff), fp)) {
		perror(filename);
		exit(1);
	}

	if (buff[0] != 'P' || buff[1] != '6') {
		fprintf(stderr, "Invalid image format (must be 'P6')\n");
		exit(1);
	}

	img = (PPMImage *) malloc(sizeof(PPMImage));
	if (!img) {
		fprintf(stderr, "Unable to allocate memory\n");
		exit(1);
	}

	c = getc(fp);
	while (c == '#') {
		while (getc(fp) != '\n');
		c = getc(fp);
	}

	ungetc(c, fp);
	if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
		fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
		exit(1);
	}

	if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
		fprintf(stderr, "Invalid rgb component (error loading '%s')\n",
				filename);
		exit(1);
	}

	if (rgb_comp_color != RGB_COMPONENT_COLOR) {
		fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
		exit(1);
	}

	while (fgetc(fp) != '\n');
	img->data = (PPMPixel*) malloc(img->x * img->y * sizeof(PPMPixel));

	if (!img) {
		fprintf(stderr, "Unable to allocate memory\n");
		exit(1);
	}

	if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
		fprintf(stderr, "Error loading image '%s'\n", filename);
		exit(1);
	}

	fclose(fp);
	return img;
}

__global__ void count_hist(PPMImage *image, float *h, float n){
	// compute i, j, k, l, x
	unsigned int index = DIM_BLOCO*DIM_BLOCO*(DIM_GRID*blockIdx.x+blockIdx.y+blockDim.y*threadIdx.x+threadIdx.y;
	unsigned int x = index/64;
	unsigned int i = index%n;
	int j = x/16;
	int k = (x-16*j)/4;
	int l = (x-16*j-4*k);
	if (index < 64*image->y*image->x
	 && image->data[i].red == j
	 && image->data[i].green == k
	 && image->data[i].blue == l) {
		atomicAdd(h[x],1/n);
	}
}

void Histogram(PPMImage *image, float *h) {
	int i, j, k, l, x, count;
	int rows, cols;
	float n = image->y * image->x;
	cols = image->x;
	rows = image->y;
	//printf("%d, %d\n", rows, cols );
	for (i = 0; i < n; i++) {
		image->data[i].red = floor((image->data[i].red * 4) / 256);
		image->data[i].blue = floor((image->data[i].blue * 4) / 256);
		image->data[i].green = floor((image->data[i].green * 4) / 256);
	}
	unsigned int size = sizeof(PPMPixel)*image->y*image->x + 2*sizeof(int);
	PPMImage *d_image;
	float *d_h;
	hipMalloc((void **)&d_image,size);
	hipMalloc((void **)&d_h,64*sizeof(float));
	hipMemcpy(d_image,image,size,hipMemcpyHostToDevice);
	hipMemcpy(d_h,h,64*sizeof(float),hipMemcpyHostToDevice);
	dim3 dimGrid(DIM_GRID,DIM_GRID);
	dim3 dimBlock(DIM_BLOCO,DIM_BLOCO);
	count_hist<<<dimGrid,dimBlock>>>(d_image,d_h,n);
	hipMemcpy(h,d_h,64*sizeof(float),hipMemcpyDeviceToHost);
	hipFree(d_image); hipFree(d_h);
	/*count = 0;
	x = 0;
	for (j = 0; j <= 3; j++) {
		for (k = 0; k <= 3; k++) {
			for (l = 0; l <= 3; l++) {
				for (i = 0; i < n; i++) {
					if (image->data[i].red == j
					 && image->data[i].green == k
					 && image->data[i].blue == l) {
						count++;
					}
				}
				h[x] = count / n; //Histograma normalizado
				count = 0;
				x++;
			}				
		}
	}*/
}

int main(int argc, char *argv[]) {
	if( argc != 2 ) {
		printf("Too many or no one arguments supplied.\n");
	}

	double t_start, t_end;
	int i;
	char *filename = argv[1]; //Recebendo o arquivo!;
	
	//scanf("%s", filename);
	PPMImage *image = readPPM(filename);

	float *h = (float*)malloc(sizeof(float) * 64);

	//Inicializar h
	for(i=0; i < 64; i++) h[i] = 0.0;

	t_start = rtclock();
	Histogram(image, h);
	t_end = rtclock();

	for (i = 0; i < 64; i++){
		printf("%0.3f ", h[i]);
	}
	printf("\n");
	//fprintf(stdout, "\n%0.6lfs\n", t_end - t_start);  
	free(h);
}

