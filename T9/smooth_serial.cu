#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#define MASK_WIDTH 5
#define COMMENT "Histogram_GPU"
#define RGB_COMPONENT_COLOR 255
#define DIM_BLOCO 32

typedef struct {
    unsigned char red, green, blue;
} PPMPixel;

typedef struct {
    int x, y;
    PPMPixel *data;
} PPMImage;

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}

static PPMImage *readPPM(const char *filename) {
    char buff[16];
    PPMImage *img;
    FILE *fp;
    int c, rgb_comp_color;
    fp = fopen(filename, "rb");
    if (!fp) {
        fprintf(stderr, "Unable to open file '%s'\n", filename);
        exit(1);
    }

    if (!fgets(buff, sizeof(buff), fp)) {
        perror(filename);
        exit(1);
    }

    if (buff[0] != 'P' || buff[1] != '6') {
        fprintf(stderr, "Invalid image format (must be 'P6')\n");
        exit(1);
    }

    img = (PPMImage *) malloc(sizeof(PPMImage));
    if (!img) {
        fprintf(stderr, "Unable to allocate memory\n");
        exit(1);
    }

    c = getc(fp);
    while (c == '#') {
        while (getc(fp) != '\n')
            ;
        c = getc(fp);
    }

    ungetc(c, fp);
    if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
        fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
        exit(1);
    }

    if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
        fprintf(stderr, "Invalid rgb component (error loading '%s')\n",
                filename);
        exit(1);
    }

    if (rgb_comp_color != RGB_COMPONENT_COLOR) {
        fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
        exit(1);
    }

    while (fgetc(fp) != '\n')
        ;
    img->data = (PPMPixel*) malloc(img->x * img->y * sizeof(PPMPixel));

    if (!img) {
        fprintf(stderr, "Unable to allocate memory\n");
        exit(1);
    }

    if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
        fprintf(stderr, "Error loading image '%s'\n", filename);
        exit(1);
    }

    fclose(fp);
    return img;
}

void writePPM(PPMImage *img) {

    fprintf(stdout, "P6\n");
    fprintf(stdout, "# %s\n", COMMENT);
    fprintf(stdout, "%d %d\n", img->x, img->y);
    fprintf(stdout, "%d\n", RGB_COMPONENT_COLOR);

    fwrite(img->data, 3 * img->x, img->y, stdout);
    fclose(stdout);
}

/* CUDA kernel */
__global__ void smoothing_gpu(PPMPixel *data, PPMPixel *data_copy, int dim_x, int dim_y){
    unsigned int index;
    int i, j, x, y;
    int total_red, total_blue, total_green;
    index = blockDim.x*blockDim.y*(gridDim.x*blockIdx.y+blockIdx.x)+blockDim.x*threadIdx.y+threadIdx.x;
    if(index < dim_x*dim_y){
        i = index/dim_x;
        j = index%dim_x;
        total_red = total_blue = total_green = 0;
        for (y = i - ((MASK_WIDTH-1)/2); y <= (i + ((MASK_WIDTH-1)/2)); y++) {
            for (x = j - ((MASK_WIDTH-1)/2); x <= (j + ((MASK_WIDTH-1)/2)); x++) {
                if (x >= 0 && y >= 0 && y < dim_y && x < dim_x) {
                    total_red += data_copy[(y * dim_x) + x].red;
                    total_blue += data_copy[(y * dim_x) + x].blue;
                    total_green += data_copy[(y * dim_x) + x].green;
                }
            }
        }
        data[(i * dim_x) + j].red = total_red / (MASK_WIDTH*MASK_WIDTH);
        data[(i * dim_x) + j].blue = total_blue / (MASK_WIDTH*MASK_WIDTH);
        data[(i * dim_x) + j].green = total_green / (MASK_WIDTH*MASK_WIDTH);
    }
}
/* End of CUDA kernel */

void Smoothing_CPU_Serial(PPMImage *image, PPMImage *image_copy) {
    int i, j, y, x;
    int total_red, total_blue, total_green;

    for (i = 0; i < image->y; i++) {
        for (j = 0; j < image->x; j++) {
            total_red = total_blue = total_green = 0;
            for (y = i - ((MASK_WIDTH-1)/2); y <= (i + ((MASK_WIDTH-1)/2)); y++) {
                for (x = j - ((MASK_WIDTH-1)/2); x <= (j + ((MASK_WIDTH-1)/2)); x++) {
                    if (x >= 0 && y >= 0 && y < image->y && x < image->x) {
                        total_red += image_copy->data[(y * image->x) + x].red;
                        total_blue += image_copy->data[(y * image->x) + x].blue;
                        total_green += image_copy->data[(y * image->x) + x].green;
                    } //if
                } //for z
            } //for y
            image->data[(i * image->x) + j].red = total_red / (MASK_WIDTH*MASK_WIDTH);
            image->data[(i * image->x) + j].blue = total_blue / (MASK_WIDTH*MASK_WIDTH);
            image->data[(i * image->x) + j].green = total_green / (MASK_WIDTH*MASK_WIDTH);
        }
    }
}

int main(int argc, char *argv[]) {

    if( argc != 2 ) {
        printf("Too many or no one arguments supplied.\n");
    }

    double t_start, t_end;
    int i;
    char *filename = argv[1]; //Recebendo o arquivo!;

    PPMImage *image = readPPM(filename);
    PPMImage *image_output = readPPM(filename);

    t_start = rtclock();
    /* CUDA stuff */
    unsigned int n = image->x*image->y;
    unsigned int dim_grid = (int)(sqrt(n*1.0/(DIM_BLOCO*DIM_BLOCO)))+1;
    unsigned int data_size = 3*(sizeof(unsigned char))*n;

    PPMPixel *d_data, *d_data_copy;
    hipMalloc((void **)&d_data, data_size);
    hipMalloc((void **)&d_data_copy, data_size);
    //
    hipMemcpy(d_data, image_output->data, data_size, hipMemcpyHostToDevice);
    hipMemcpy(d_data_copy, image->data, data_size, hipMemcpyHostToDevice);
    //
    dim3 dimBlock(DIM_BLOCO,DIM_BLOCO);
    dim3 dimGrid(dim_grid,dim_grid);
    //
    smoothing_gpu<<<dimGrid,dimBlock>>>(d_data, d_data_copy, image->x, image->y);
    //
    hipMemcpy(image_output->data, d_data, data_size, hipMemcpyDeviceToHost);
    //
    hipFree(d_data, d_data_copy);
    /* End of CUDA stuff */
    t_end = rtclock();

    //Smoothing_CPU_Serial(image_output, image);
    writePPM(image_output);
    //fprintf(stdout, "\n%0.6lfs\n", t_end - t_start);  
    free(image);
    free(image_output);
}
