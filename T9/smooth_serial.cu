
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#define MASK_WIDTH 11
#define COMMENT "Histogram_GPU"
#define RGB_COMPONENT_COLOR 255
#define DIM_BLOCO 32

typedef struct {
    unsigned char red, green, blue;
} PPMPixel;

typedef struct {
    int x, y;
    PPMPixel *data;
} PPMImage;

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}

static PPMImage *readPPM(const char *filename) {
    char buff[16];
    PPMImage *img;
    FILE *fp;
    int c, rgb_comp_color;
    fp = fopen(filename, "rb");
    if (!fp) {
        fprintf(stderr, "Unable to open file '%s'\n", filename);
        exit(1);
    }

    if (!fgets(buff, sizeof(buff), fp)) {
        perror(filename);
        exit(1);
    }

    if (buff[0] != 'P' || buff[1] != '6') {
        fprintf(stderr, "Invalid image format (must be 'P6')\n");
        exit(1);
    }

    img = (PPMImage *) malloc(sizeof(PPMImage));
    if (!img) {
        fprintf(stderr, "Unable to allocate memory\n");
        exit(1);
    }

    c = getc(fp);
    while (c == '#') {
        while (getc(fp) != '\n')
            ;
        c = getc(fp);
    }

    ungetc(c, fp);
    if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
        fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
        exit(1);
    }

    if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
        fprintf(stderr, "Invalid rgb component (error loading '%s')\n",
                filename);
        exit(1);
    }

    if (rgb_comp_color != RGB_COMPONENT_COLOR) {
        fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
        exit(1);
    }

    while (fgetc(fp) != '\n')
        ;
    img->data = (PPMPixel*) malloc(img->x * img->y * sizeof(PPMPixel));

    if (!img) {
        fprintf(stderr, "Unable to allocate memory\n");
        exit(1);
    }

    if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
        fprintf(stderr, "Error loading image '%s'\n", filename);
        exit(1);
    }

    fclose(fp);
    return img;
}

void writePPM(PPMImage *img) {

    fprintf(stdout, "P6\n");
    fprintf(stdout, "# %s\n", COMMENT);
    fprintf(stdout, "%d %d\n", img->x, img->y);
    fprintf(stdout, "%d\n", RGB_COMPONENT_COLOR);

    fwrite(img->data, 3 * img->x, img->y, stdout);
    fclose(stdout);
}

/* CUDA kernel */
__global__ void smoothing_gpu(PPMPixel *data, PPMPixel *data_copy, int dim_x, int dim_y){
    unsigned int index, index_in_block;
    int pos0_x, pos0_y; // A posição na imagem do primeiro pixel do bloco
    int img_x, img_y; // A posição do pixel da thread na imagem
    //
    int x, y, lx, ly;
    int total_red, total_blue, total_green;
    //
    pos0_x = blockIdx.x*(blockDim.x-2*(MASK_WIDTH/2));
    pos0_y = blockIdx.y*(blockDim.y-2*(MASK_WIDTH/2));
    img_x = pos0_x+(threadIdx.x-MASK_WIDTH/2);
    img_y = pos0_y+(threadIdx.y-MASK_WIDTH/2);
    index_in_block = blockDim.x*threadIdx.y+threadIdx.x;
    //if(img_x < dim_x && img_x >= 0 && img_y < dim_y && img_y >= 0){
    PPMPixel s_data;
    __shared__ PPMPixel s_data_copy[(DIM_BLOCO+2*(MASK_WIDTH/2))*(DIM_BLOCO+2*(MASK_WIDTH/2))];
	if(img_x < dim_x && img_x >= 0 && img_y < dim_y && img_y >= 0){
    	s_data_copy[index_in_block] = data_copy[img_y*dim_x+img_x];
	} else {
		s_data_copy[index_in_block].red = 0;
		s_data_copy[index_in_block].blue = 0;
		s_data_copy[index_in_block].green = 0;
	}
    __syncthreads();
	if(img_x-pos0_x < DIM_BLOCO
	&& img_x-pos0_x >= 0
	&& img_y-pos0_y < DIM_BLOCO
	&& img_y-pos0_y >= 0
	&& img_x < dim_x
	&& img_x >= 0
	&& img_y < dim_y
	&& img_y >= 0){
		total_red = total_blue = total_green = 0;
		for (y = img_y - ((MASK_WIDTH-1)/2); y <= (img_y + ((MASK_WIDTH-1)/2)); y++) {
		    for (x = img_x - ((MASK_WIDTH-1)/2); x <= (img_x + ((MASK_WIDTH-1)/2)); x++) {
		        //if (x >= 0 && y >= 0 && y < dim_y && x < dim_x) {
		            lx = x - img_x + threadIdx.x;
		            ly = y - img_y + threadIdx.y;
		            total_red += s_data_copy[ly*blockDim.x+lx].red;
		            total_blue += s_data_copy[ly*blockDim.x+lx].blue;
		            total_green += s_data_copy[ly*blockDim.x+lx].green;
		        //}
		    }
		}
		s_data.red = total_red / (MASK_WIDTH*MASK_WIDTH);
		s_data.blue = total_blue / (MASK_WIDTH*MASK_WIDTH);
		s_data.green = total_green / (MASK_WIDTH*MASK_WIDTH);
		data[img_y*dim_x+img_x] = s_data;
    }
}
/* End of CUDA kernel */

/* void Smoothing_CPU_Serial(PPMImage *image, PPMImage *image_copy) {
    int i, j, y, x;
    int total_red, total_blue, total_green;

    for (i = 0; i < image->y; i++) {
        for (j = 0; j < image->x; j++) {
            total_red = total_blue = total_green = 0;
            for (y = i - ((MASK_WIDTH-1)/2); y <= (i + ((MASK_WIDTH-1)/2)); y++) {
                for (x = j - ((MASK_WIDTH-1)/2); x <= (j + ((MASK_WIDTH-1)/2)); x++) {
                    if (x >= 0 && y >= 0 && y < image->y && x < image->x) {
                        total_red += image_copy->data[(y * image->x) + x].red;
                        total_blue += image_copy->data[(y * image->x) + x].blue;
                        total_green += image_copy->data[(y * image->x) + x].green;
                    } //if
                } //for z
            } //for y
            image->data[(i * image->x) + j].red = total_red / (MASK_WIDTH*MASK_WIDTH);
            image->data[(i * image->x) + j].blue = total_blue / (MASK_WIDTH*MASK_WIDTH);
            image->data[(i * image->x) + j].green = total_green / (MASK_WIDTH*MASK_WIDTH);
        }
    }
} */

int main(int argc, char *argv[]) {

    if( argc != 2 ) {
        printf("Too many or no one arguments supplied.\n");
    }

    double t_start, t_end;
    int i;
    char *filename = argv[1]; //Recebendo o arquivo!;
    //int MASK_WIDTH;
    //scanf("%d",&MASK_WIDTH);

    PPMImage *image = readPPM(filename);
    PPMImage *image_output = readPPM(filename);

    t_start = rtclock();
    /* CUDA stuff */
    unsigned int n = image->x*image->y;
    unsigned int dim_grid_x = (image->x+DIM_BLOCO)/DIM_BLOCO;
    unsigned int dim_grid_y = (image->y+DIM_BLOCO)/DIM_BLOCO;
    unsigned int data_size = 3*(sizeof(unsigned char))*n;

    PPMPixel *d_data, *d_data_copy;
    hipMalloc((void **)&d_data, data_size);
    hipMalloc((void **)&d_data_copy, data_size);
    //
    hipMemcpy(d_data, image_output->data, data_size, hipMemcpyHostToDevice);
    hipMemcpy(d_data_copy, image->data, data_size, hipMemcpyHostToDevice);
    //
    dim3 dimBlock(DIM_BLOCO+2*(MASK_WIDTH/2),DIM_BLOCO+2*(MASK_WIDTH/2));
    dim3 dimGrid(dim_grid_x,dim_grid_y);
    //
    smoothing_gpu<<<dimGrid,dimBlock>>>(d_data, d_data_copy, image->x, image->y);
    //
    hipMemcpy(image_output->data, d_data, data_size, hipMemcpyDeviceToHost);
    //
    hipFree(d_data); hipFree(d_data_copy);
    /* End of CUDA stuff */
    t_end = rtclock();

    writePPM(image_output);
    //fprintf(stdout, "\n%0.6lfs\n", t_end - t_start);  
    free(image);
    free(image_output);
}
